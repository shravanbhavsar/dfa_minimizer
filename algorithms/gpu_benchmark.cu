// gpu_benchmark.cu

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// CUDA kernel: mark mask[i]=1 if state i has a transition into `splitter`
__global__ void compute_target_partitions(
    const int *T,    // flattened n×k transition table
    const int *part, // partition IDs array, length n
    int n,           // number of states
    int k,           // alphabet size
    int splitter,    // partition ID to test against
    int *mask        // output mask, length n
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    int m = 0;
    for (int j = 0; j < k; ++j) {
        int tgt = T[idx*k + j];
        if (part[tgt] == splitter) { m = 1; break; }
    }
    mask[idx] = m;
}

// CUDA kernel: for each i with part[i]==q && mask[i]==1, assign part[i]=new_q
__global__ void split_partition(
    int *part,
    const int *mask,
    int n,
    int q,
    int new_q,
    int *flag      // device flag: set to 1 if any split occurred
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    if (part[idx] == q && mask[idx]) {
        part[idx] = new_q;
        *flag = 1;
    }
}

// Host function: GPU-based partition refinement
void gpu_minimize(int *h_T, int *h_part, int n, int k) {
    int *d_T, *d_part, *d_mask, *d_flag;
    size_t size_T = n * k * sizeof(int);
    size_t size_n = n * sizeof(int);

    hipMalloc(&d_T, size_T);
    hipMalloc(&d_part, size_n);
    hipMalloc(&d_mask, size_n);
    hipMalloc(&d_flag, sizeof(int));

    hipMemcpy(d_T, h_T, size_T, hipMemcpyHostToDevice);
    hipMemcpy(d_part, h_part, size_n, hipMemcpyHostToDevice);

    int threads = 128;
    int blocks = (n + threads - 1) / threads;

    // Worklist of partition IDs
    int *worklist = (int*)malloc((n+2)*sizeof(int));
    int wl_size = 0;
    worklist[wl_size++] = 0;  // finals
    worklist[wl_size++] = 1;  // non-finals
    int max_q = 1;

    while (wl_size > 0) {
        int splitter = worklist[--wl_size];

        // Compute mask
        compute_target_partitions<<<blocks,threads>>>(d_T, d_part, n, k, splitter, d_mask);
        hipDeviceSynchronize();

        // Try splitting each partition q = 0..max_q
        for (int q = 0; q <= max_q; ++q) {
            int zero = 0;
            hipMemcpy(d_flag, &zero, sizeof(int), hipMemcpyHostToDevice);

            int new_q = max_q + 1;
            split_partition<<<blocks,threads>>>(d_part, d_mask, n, q, new_q, d_flag);
            hipDeviceSynchronize();

            int changed;
            hipMemcpy(&changed, d_flag, sizeof(int), hipMemcpyDeviceToHost);
            if (changed) {
                // Pull part back to decide smaller piece
                int *temp = (int*)malloc(size_n);
                hipMemcpy(temp, d_part, size_n, hipMemcpyDeviceToHost);
                int cnt1=0, cnt2=0;
                for (int i = 0; i < n; ++i) {
                    if (temp[i] == new_q) ++cnt1;
                    else if (temp[i] == q) ++cnt2;
                }
                free(temp);
                worklist[wl_size++] = (cnt1 < cnt2 ? new_q : q);
                max_q = new_q;
            }
        }
    }

    hipMemcpy(h_part, d_part, size_n, hipMemcpyDeviceToHost);

    free(worklist);
    hipFree(d_T);
    hipFree(d_part);
    hipFree(d_mask);
    hipFree(d_flag);
}

int main() {
    // Benchmark parameters
    int sizes[] = {100, 500, 1000};
    int numSizes = sizeof(sizes)/sizeof(sizes[0]);
    int trials = 3;
    int k = 2;  // binary alphabet
    srand((unsigned)time(NULL));

    for (int si = 0; si < numSizes; ++si) {
        int n = sizes[si];
        float total_ms = 0.0f;

        for (int t = 0; t < trials; ++t) {
            // Allocate host arrays
            int *h_T    = (int*)malloc(n * k * sizeof(int));
            int *h_part = (int*)malloc(n * sizeof(int));

            // 1) Random transitions
            for (int i = 0; i < n; ++i) {
                for (int j = 0; j < k; ++j) {
                    h_T[i*k + j] = rand() % n;
                }
            }
            // 2) Random final‐state subset
            int numF = rand() % n + 1;
            int *idxs = (int*)malloc(n * sizeof(int));
            for (int i = 0; i < n; ++i) idxs[i] = i;
            for (int i = 0; i < numF; ++i) {
                int r = i + rand() % (n - i);
                int tmp = idxs[i]; idxs[i]=idxs[r]; idxs[r]=tmp;
            }
            // initial partition: 0=final, 1=non-final
            for (int i = 0; i < n; ++i) h_part[i] = 1;
            for (int i = 0; i < numF; ++i) h_part[idxs[i]] = 0;
            free(idxs);

            // Measure GPU minimization time
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);

            gpu_minimize(h_T, h_part, n, k);

            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float ms = 0;
            hipEventElapsedTime(&ms, start, stop);
            total_ms += ms;

            hipEventDestroy(start);
            hipEventDestroy(stop);

            free(h_T);
            free(h_part);
        }

        printf("n = %4d : GPU avg time = %8.3f ms\n", n, total_ms / trials);
    }
    return 0;
}
